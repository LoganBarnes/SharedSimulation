#include "hip/hip_runtime.h"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "hip/hip_runtime_api.h"
#include "helper_grid.h"


extern "C"
{
  /**
   * @brief setup_kernel
   * @param state
   * @param seed
   */
  __global__
  void
  initCuRand_kernel(
                    hiprandState *state,
                    uint64_t     offset,
                    uint64_t     seed,
                    dim3         texDim
                    )
  {
    uint x = blockIdx.x * blockDim.x + threadIdx.x;
    uint y = blockIdx.y * blockDim.y + threadIdx.y;

    if ( x < texDim.x && y < texDim.y )
    {
      uint id = y * texDim.x + x;
      hiprand_init( seed, id, 0, &state[ id ] + offset );
    }
  }



  /**
   * @brief cuda_LinitCuRand
   * @param state
   * @param seed
   */
  void
  cuda_initCuRand(
                  hiprandState *state,
                  uint64_t     offset,
                  uint64_t     seed,
                  dim3         texDim
                  )
  {
    dim3 thread( 32, 32 );
    dim3 block( 1 );

    computeGridSize( texDim.x, thread.x, block.x, thread.x );
    computeGridSize( texDim.y, thread.y, block.y, thread.y );

    initCuRand_kernel << < block, thread >> > ( state, offset, seed, texDim );
  }



}
