#if defined( WIN32 ) || defined( _WIN32 ) || defined( WIN64 ) || defined( _WIN64 )
#include <windows.h>
#endif
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <hip/hip_runtime_api.h>
#include "hip/hip_runtime_api.h"
#include <iostream>

extern "C"
{

void
cuda_init(
          const int    argc,
          const char **argv,
          const bool   print
          )
{
  int devID;

  // use device with highest Gflops/s
  devID = findCudaDevice( argc, argv, print );

  if ( devID < 0 )
  {
    printf( "No CUDA Capable devices found, exiting...\n" );
    exit( EXIT_SUCCESS );
  }
  std::cout << std::flush;
}



void
cuda_destroy( const bool print )
{
  // hipDeviceReset causes the driver to clean up all state. While
  // not mandatory in normal operation, it is good practice.  It is also
  // needed to ensure correct operation when the application is being
  // profiled. Calling hipDeviceReset causes all profile data to be
  // flushed before the application exits
  hipDeviceReset( );

  if ( print )
  {
    std::cout << "Cuda device reset" << std::endl;
  }
}



void
cuda_malloc(
            void **devPtr,
            size_t size
            )
{
  checkCudaErrors( hipMalloc( devPtr, size ) );
}



void
cuda_free( void *devPtr )
{
  checkCudaErrors( hipFree( devPtr ) );
}



void
cuda_memcpy(
            void               *dst,
            const void         *src,
            size_t              count,
            enum hipMemcpyKind kind
            )
{
  checkCudaErrors( hipMemcpy( dst, src, count, kind ) );
}



void
cuda_graphicsGLRegisterImage(
                             hipGraphicsResource_t   *resource,
                             GLuint                    tex,
                             GLenum                    target,
                             hipGraphicsRegisterFlags flags
                             )
{
  checkCudaErrors( hipGraphicsGLRegisterImage( resource, tex, target, flags ) );
}



void
cuda_graphicsUnregisterResource( hipGraphicsResource_t resource )
{
  checkCudaErrors( hipGraphicsUnregisterResource( resource ) );
}



void
cuda_graphicsMapResources( hipGraphicsResource_t *res )
{
  checkCudaErrors( hipGraphicsMapResources( 1, res ) );
}



void
cuda_graphicsUnmapResources( hipGraphicsResource_t *res )
{
  checkCudaErrors( hipGraphicsUnmapResources( 1, res ) );
}



void
cuda_graphicsSubResourceGetMappedArray(
                                       hipArray_t           *array,
                                       hipGraphicsResource_t res,
                                       GLuint                 index,
                                       GLuint                 level
                                       )
{
  checkCudaErrors( hipGraphicsSubResourceGetMappedArray( array, res, index, level ) );
}



void
cuda_createSurfaceObject(
                         hipSurfaceObject_t *surface,
                         hipResourceDesc    *desc
                         )
{
  checkCudaErrors( hipCreateSurfaceObject( surface, desc ) );
}



void
cuda_destroySurfaceObject( hipSurfaceObject_t surface )
{
  checkCudaErrors( hipDestroySurfaceObject( surface ) );
}



void
cuda_streamSynchronize( hipStream_t stream )
{
  checkCudaErrors( hipStreamSynchronize( stream ) );
}



void
cuda_deviceSynchronize( )
{
  checkCudaErrors( hipDeviceSynchronize( ) );
}



void
cuda_profilerStart( )
{
  checkCudaErrors( hipProfilerStart( ) );
}



void
cuda_profilerStop( )
{
  checkCudaErrors( hipProfilerStop( ) );
}



} // extern "C"
