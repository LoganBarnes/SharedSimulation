#if defined(WIN32) || defined(_WIN32) || defined(WIN64) || defined(_WIN64)
#include <windows.h>
#endif
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <hip/hip_runtime_api.h>
#include "hip/hip_runtime_api.h"

typedef unsigned int uint;
typedef unsigned long ulong;

extern "C"
{
  void
  cuda_init(
            int          argc,
            const char **argv,
            bool         print
            )
  {
    int devID;

    // use device with highest Gflops/s
    devID = findCudaDevice( argc, argv, print );

    if ( devID < 0 )
    {
      printf( "No CUDA Capable devices found, exiting...\n" );
      exit( EXIT_SUCCESS );
    }
  }



  void
  cuda_destroy( )
  {
    // hipDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling hipDeviceReset causes all profile data to be
    // flushed before the application exits
    hipDeviceReset( );
  }



  void
  cuda_allocateArray(
                     void **devPtr,
                     size_t size
                     )
  {
    checkCudaErrors( hipMalloc( devPtr, size ) );
  }



  void
  cuda_freeArray( void *devPtr )
  {
    checkCudaErrors( hipFree( devPtr ) );
  }



  void
  cuda_copyArrayToDevice(
                         void       *device,
                         const void *host,
                         size_t      offset,
                         size_t      size
                         )
  {
    checkCudaErrors( hipMemcpy( ( char* ) device + offset, host, size, hipMemcpyHostToDevice ) );
  }



  void
  cuda_copyArrayFromDevice(
                           void       *host,
                           const void *device,
                           size_t      size
                           )
  {
    checkCudaErrors( hipMemcpy( host, device, size, hipMemcpyDeviceToHost ) );
  }



  void
  cuda_registerGLTexture(
                         hipGraphicsResource_t   *resource,
                         GLuint                    tex,
                         GLenum                    target,
                         hipGraphicsRegisterFlags flags
                         )
  {
    checkCudaErrors( hipGraphicsGLRegisterImage( resource, tex, target, flags ) );
  }



  void
  cuda_unregisterResource( hipGraphicsResource_t resource )
  {
    checkCudaErrors( hipGraphicsUnregisterResource( resource ) );
  }



  void
  cuda_graphicsMapResource( hipGraphicsResource_t *res )
  {
    checkCudaErrors( hipGraphicsMapResources( 1, res ) );
  }



  void
  cuda_graphicsUnmapResource( hipGraphicsResource_t *res )
  {
    checkCudaErrors( hipGraphicsUnmapResources( 1, res ) );
  }



  void
  cuda_graphicsSubResourceGetMappedArray(
                                         hipArray_t           *array,
                                         hipGraphicsResource_t res,
                                         GLuint                 index,
                                         GLuint                 level
                                         )
  {
    checkCudaErrors( hipGraphicsSubResourceGetMappedArray( array, res, index, level ) );
  }



  void
  cuda_createSurfaceObject(
                           hipSurfaceObject_t *surface,
                           hipResourceDesc    *desc
                           )
  {
    checkCudaErrors( hipCreateSurfaceObject( surface, desc ) );
  }



  void
  cuda_destroySurfaceObject( hipSurfaceObject_t surface )
  {
    checkCudaErrors( hipDestroySurfaceObject( surface ) );
  }



  void
  cuda_streamSynchronize( hipStream_t stream )
  {
    checkCudaErrors( hipStreamSynchronize( stream ) );
  }



  void
  cuda_deviceSynchronize( )
  {
    checkCudaErrors( hipDeviceSynchronize( ) );
  }



  void
  cuda_profilerStart( )
  {
    checkCudaErrors( hipProfilerStart( ) );
  }



  void
  cuda_profilerStop( )
  {
    checkCudaErrors( hipProfilerStop( ) );
  }



}
